#include "hip/hip_runtime.h"
#include "../inc/Renderer.h"

#include <cstdint>

#include "../inc/BFSJob.cuh"
#include "../inc/bfsoctree_operations.h"
#include "../inc/Light.h"

// Include the implementations of all math functions.
// CUDA requires that function declarations and definitions are
// in the same .cu file.
#include "math3d.cpp"

/* Textures */
static texture< unsigned, hipTextureType1D, hipReadModeElementType > tDepthBuffer;
static texture< uchar4, hipTextureType2D, hipReadModeNormalizedFloat > tDiffuse;
static texture< uchar4, hipTextureType2D, hipReadModeNormalizedFloat > tIllum;
static texture< uchar4, hipTextureType2D, hipReadModeNormalizedFloat > tSpec;
static texture< uchar4, hipTextureType2D, hipReadModeNormalizedFloat > tNormal;

unsigned long int d_getChildCountFromMask( unsigned long int mask )
{
    return (   1ul & mask ) +
          ((   2ul & mask ) >> 1 ) +
          ((   4ul & mask ) >> 2 ) +
          ((   8ul & mask ) >> 3 ) +
          ((  16ul & mask ) >> 4 ) +
          ((  32ul & mask ) >> 5 ) +
          ((  64ul & mask ) >> 6 ) +
          (( 128ul & mask ) >> 7 );
}

/**
 * The main kernel responsible for rendering. Equivalent to the rasterizer plus vertex shader.
 * The kernel is invoked with as many threads as the job queue contains elements.
 * Every thread processes one job and either draws the voxel that blongs ot it or
 * decides that the voxel needs further subdivision and adds new jobs to the queue
 * (one for every child of the voxel).
 *
 * In the first case, the voxel isn't actually drawn but rather its depth and
 * visual data is stored in the depth map and voxel buffer at the coordinates that
 * correspond to the voxel's center. This is done because it is cheaper to scan multiple
 * neighboring pixels from the depth map for a voxel than it is to write a voxel's depth into
 * multiple pixels of the depth map (which requires multiple atomicMin calls).
 *
 * Drawing a voxel consists of projecting it into screen space and determining visibility
 * through a depth buffer. The actual coloring happens at a later stage.
 *
 * @param innerNodeCount The number of inner nodes of the octree that is traversed.
 * @param innerNodes     The inner nodes of the octree.
 * @param leaves         The leaf nodes of the octree.
 * @param dimension      The dimension of the octree (cube).
 * @param world          The world transformation of the model the octree represents.
 * @param view           The view transformation of the virtual camera.
 * @param projection     The projection transformation of the virtual camera.
 * @param animation      A set of matrices encoding the bone transformations for every frame.
 * @param boneCount      The number of bones of the model.
 * @param depthBuffer    The depth buffer.
 * @param voxelBuffer    The voxel buffer (think input to the pixel/fragment shader).
 */
static __global__ void traverse
(
	unsigned long int innerNodeCount,
    BFSInnerNode * innerNodes,
    BFSLeaf * leaves,
    float dimension,
    Matrix world, Vector3 camPos, Matrix view, Matrix projection,
    Matrix * animation, unsigned char boneCount,
    unsigned int * depthBuffer, VoxelData * voxelBuffer,
	int frameWidth, int frameHeight,
	int animationFrameIndex,
	int octreeLevel,
	// TODO: Rename
	int const * startIndex, int const * endIndex,
	int * travQueuePtr,
	BFSJob * jobQueue
)
{
	unsigned long int index = blockDim.x * blockIdx.x + threadIdx.x + ( * startIndex );	
	short int x, y, z, w = 2;
	unsigned int depth;

	__shared__ unsigned int sharedTravQueuePtr;

	if (threadIdx.x == 0)
		sharedTravQueuePtr = 0;

	__syncthreads();

	if( index < ( * endIndex ) )
	{
		BFSJob job = jobQueue[ index ];
		BFSInnerNode node;
		node.childPtr = 0;
		node.mask = 0;
		
		if (job.index < innerNodeCount)
			node = innerNodes[job.index];
		else
			node.vd = leaves[job.index - innerNodeCount].vd;
			
		float gridCellDim = dimension / ( (float) ( 1 << octreeLevel ) );
		float gridCellHalfDim = gridCellDim * 0.5f;
		
		float minCoord = -0.5f * dimension;
		Vector3 center = { fmaf(job.x, gridCellDim, minCoord + gridCellHalfDim),
					       fmaf(job.y, gridCellDim, minCoord + gridCellHalfDim),
						   fmaf(job.z, gridCellDim, minCoord + gridCellHalfDim) };		
		
		Vector3 skinnedCenter = ZERO;
		unsigned char involvedBones = 0;
        if (node.vd.boneWeights.x > 0.f)
		{
			skinnedCenter = d_vecMulS
			(
				d_vecMulM(center, animation[ animationFrameIndex * boneCount+node.vd.boneIndex0 ]),
				node.vd.boneWeights.x
			);
			++involvedBones;
		}
		if (node.vd.boneWeights.y > 0.f)
		{
			skinnedCenter = d_vecAddVec( skinnedCenter, d_vecMulS(
				d_vecMulM(center, animation[ animationFrameIndex * boneCount+node.vd.boneIndex1 ]), node.vd.boneWeights.y
			));
			++involvedBones;
		}
		if (node.vd.boneWeights.z > 0.f)
		{
			skinnedCenter = d_vecAddVec( skinnedCenter, d_vecMulS(
				d_vecMulM(center, animation[ animationFrameIndex * boneCount+node.vd.boneIndex2 ]), node.vd.boneWeights.z
			));
			++involvedBones;
		}
		if (node.vd.boneWeights.w > 0.f)
		{
			skinnedCenter = d_vecAddVec(skinnedCenter, d_vecMulS(
				d_vecMulM(center, animation[ animationFrameIndex * boneCount+node.vd.boneIndex3 ]), node.vd.boneWeights.w
			));
			++involvedBones;
		}
		center = skinnedCenter;
		Vector3 originalCenter = center;

		// resizing of voxel to counter stretching.
		gridCellHalfDim *= fminf(2.f, involvedBones);

		center = d_vecMulM(center, world);
		Vector3 eyeVec = d_vecNormalize(d_vecSubVec(camPos, center));

		center = d_vecMulM(center, view);
		Vector3 dimVec = { gridCellHalfDim, gridCellHalfDim, center.z };
		
		center = d_vecMulM(center, projection);
		dimVec = d_vecMulM(dimVec, projection);
							
		//viewing frustum + clockwise culling
		node.vd.normal = d_vecMulM(node.vd.normal, world);
		// TODO: Add check whether adding the current node's children
		// to the job queue would exceed the queue's size limit
		if ( -1.f <= center.x + dimVec.x && center.x - dimVec.x <= 1.f &&
			-1.f <= center.y + dimVec.x && center.y - dimVec.x <= 1.f &&
			 0.f <= center.z + dimVec.x && center.z - dimVec.x <= 1.f &&
			 ( octreeLevel <= 8 || d_vecDot( node.vd.normal, eyeVec ) >= -0.4f ))
		{	
			center.x = (center.x + 1.f) * frameWidth * 0.5f;
			center.y = frameHeight - (center.y + 1.f) * frameHeight * 0.5f;

			dimVec.x *= frameWidth;
			dimVec.y *= frameHeight;

			x = center.x;
			y = center.y;				

			x = max(x, 0);
			x = min(x, frameWidth - 1);
			y = max(y, 0);
			y = min(y, frameHeight - 1);		
								
			depth = center.z * INT_MAX;
			index = x + y * frameWidth;

			if ((dimVec.x > 1.f) && (z = d_getChildCountFromMask(node.mask)) != 0)
			{	
				index = atomicAdd(&sharedTravQueuePtr, z);
				w = 0;
			}
			else
			{
				atomicMin(depthBuffer + index, depth);
				w = 1;
			}
		}
		
		__syncthreads();
		if( threadIdx.x == 0 )
		{
			sharedTravQueuePtr = atomicAdd( travQueuePtr, sharedTravQueuePtr );
		}

		__syncthreads();
			
		if (w == 0)
		{
			index += sharedTravQueuePtr;

#pragma unroll 8
			for (w = 0; w < 8; ++w)
			{
				if ((node.mask & (1u << w)) != 0)
				{
					x = 2 * job.x + (w & 1u);
					y = 2 * job.y + ((w & 2u) >> 1);
					z = 2 * job.z + ((w & 4u) >> 2);

					jobQueue[ index++ ] = dmake_BFSJob( node.childPtr++, x, y, z );
				}
			}				
		}
		else if (w == 1 && depth == depthBuffer[index])
		{
			VoxelData vd = { node.vd.normal,
							 d_vecMulM(node.vd.tangent, world),
							 center.x,
							 center.y,
							 dimVec.x,
							 dimVec.y,
							 node.vd.texCoord,
							 eyeVec,
							 originalCenter };

			voxelBuffer[index] = vd;		
		}
	}
}

/**
 * Draws an image of a rendered voxel model to the screen. For every pixel p visible
 * on the screen it scans a certain number of neighboring pixels in the depth map for a voxel and
 * selects the nearest voxel that covers p.
 *
 * This function could be easily implemented in the form of a shader by
 * storing the voxel data in a set of textures (one for every voxel property like
 * normals, texCoords, etc.) and send them to the GPU, which could combine this data with
 * triangle meshes (since one can output depth information in pixel/fragment shaders).
 *
 * @param depthBuffer              The depth buffer.
 * @param colorBuffer              The color buffer.
 * @param voxelBuffer              The voxel buffer.
 * @param shadowMap                The shadow map.
 * @param light                    The light direction.
 * @param lightWorldViewProjection light transform * model world transform * camera view transform * camera projection transform
 * @param diffusPower              The diffuse intensity of the light source.
 */
static __global__ void draw
(
	unsigned int * depthBuffer,
	uchar4 * colorBuffer,
	VoxelData * voxelBuffer,
	float * shadowMap,
	int frameWidth, int frameHeight,

    Vector3 light,
	Matrix lightWorldViewProjection,
	float diffusePower
)
{
	unsigned index = blockIdx.x * blockDim.x + threadIdx.x, index2;
	unsigned minDepth = INT_MAX;
	unsigned depth;
	int startIndex, curIndex, x, y;
	VoxelData vd, minVd;

	if( index < frameWidth * frameHeight )
	{
		y = index / frameWidth;
		x = index - y * frameWidth;
		
		startIndex = index - frameWidth - 1;
		curIndex = 0;

#pragma unroll 9
		for (int i = 0; i < 9; ++i)
		{
			index2 = min(max(startIndex + curIndex, 0), frameWidth * frameHeight);
			if ((depth = tex1Dfetch(tDepthBuffer, index2)) < minDepth)
			{		
				vd = voxelBuffer[index2];				
				if (fabsf(vd.pos.x - .5f - x) <= vd.pos.z)			
					if (fabsf(vd.pos.y - .5f - y) <= vd.pos.w)
					{
						minDepth = depth;	
						minVd = vd;
					}			
			}

			if (++curIndex == 3)
			{
				curIndex = 0;
				startIndex += frameWidth;
			}
		}
		
		if (minDepth < INT_MAX)
		{
			float4 color = make_float4(0.f, 0.f, 0.f, 0.f);			

			//shadow mapping
			minVd.center = d_vecMulM(minVd.center, lightWorldViewProjection);
			minVd.center.x = (minVd.center.x + 1.f) * frameWidth * 0.5f;
			minVd.center.y = frameHeight - (minVd.center.y + 1.f) * frameHeight * 0.5f;

			x = minVd.center.x;
			y = minVd.center.y;				

			x = max(x, 0);
			x = min(x, frameWidth - 1);
			y = max(y, 0);
			y = min(y, frameHeight - 1);
			
			index2 = x + y * frameWidth;			
			
			float4 tempf;
			if (minVd.center.z <= shadowMap[index2] + 0.01f) //not in shadow		
			{
				//normal mapping
				tempf = tex2D(tNormal, minVd.texCoord.x, minVd.texCoord.y);
				Vector3 tempv = d_vecCross(minVd.normal, minVd.tangent);
				minVd.normal = d_vecNormalize(d_vecAddVec(d_vecMulS(minVd.normal, tempf.z),
											  d_vecAddVec(d_vecMulS(minVd.tangent, tempf.x),
														  d_vecMulS(tempv, tempf.y))));
				
				tempf = tex2D(tIllum, minVd.texCoord.x, minVd.texCoord.y);				
				float intensity = fmaxf(0.f, d_vecDot(minVd.normal, light));				
				if (intensity > 0.f || diffusePower < 1.f)
				{					
					color = tex2D(tDiffuse, minVd.texCoord.x, minVd.texCoord.y);
					color.x *= intensity * diffusePower + tempf.x + 1.0f - diffusePower;
					color.y *= intensity * diffusePower + tempf.y + 1.0f - diffusePower;
					color.z *= intensity * diffusePower + tempf.z + 1.0f - diffusePower;
				}

				tempv = d_vecNormalize(d_vecAddVec(d_vecMulS(minVd.eyeVec, 0.5f), d_vecMulS(light, 0.5f)));				
				intensity = powf(fmaxf(0.f, d_vecDot(tempv, minVd.normal)), 32.f); 
				if (intensity > 0.f)
				{
					tempf = tex2D(tSpec, minVd.texCoord.x, minVd.texCoord.y);
					color.x += diffusePower * intensity * tempf.x;
					color.y += diffusePower * intensity * tempf.y;
					color.z += diffusePower * intensity * tempf.z;
				}
			}
			else
			{
				color = tex2D(tDiffuse, minVd.texCoord.x, minVd.texCoord.y);
				tempf = tex2D(tIllum, minVd.texCoord.x, minVd.texCoord.y);				
				color.x *= tempf.x + 1.0f - diffusePower;
				color.y *= tempf.y + 1.0f - diffusePower;
				color.z *= tempf.z + 1.0f - diffusePower;
			}
					
			color.x = fminf(1.f, color.x);
			color.y = fminf(1.f, color.y);
			color.z = fminf(1.f, color.z);
			
			colorBuffer[index] = make_uchar4(color.x * 255.f, color.y * 255.f, color.z * 255.f, 0);
		}
	}
}

/**
 * @see draw
 * Like draw but outputs a shadow map.
 *
 * @param depthBuffer The depth buffer.
 * @param shadowMap   The shadow map to output the data to.
 * @param voxelBuffer The voxel buffer.
 */
static __global__ void drawShadowMap
(
	unsigned int * depthBuffer,
	float * shadowMap,
	VoxelData * voxelBuffer,
	int frameWidth, int frameHeight
)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x, index2, minDepth = INT_MAX, depth;
	int startIndex, curIndex, x, y;
	VoxelData vd;

	if (index < frameWidth * frameHeight)
	{
		y = index / frameWidth;
		x = index - y * frameWidth;
		
		startIndex = index - frameWidth - 1;
		curIndex = 0;

#pragma unroll 9
		for (int i = 0; i < 9; ++i)
		{
			index2 = min(max(startIndex + curIndex, 0), frameWidth * frameHeight);						
			if ((depth = tex1Dfetch(tDepthBuffer, index2)) < minDepth)
			{		
				vd = voxelBuffer[index2];				
				if (fabsf(vd.pos.x - .5f - x) <= vd.pos.z)				
					if (fabsf(vd.pos.y - .5f - y) <= vd.pos.w)					
						minDepth = depth;				
			}

			if (++curIndex == 3)
			{
				curIndex = 0;
				startIndex += frameWidth;
			}
		}
		
		if (minDepth < INT_MAX)
		{
			float color = ((float)minDepth) / ((float)INT_MAX);
			shadowMap[index] = color;
		}
	}
}

Renderer::Renderer( int frameWidthInPixels, int frameHeightInPixels, bool shadowMapping ) :
	m_frameWidth( frameWidthInPixels ),
	m_frameHeight( frameHeightInPixels ),
	m_shadowMapping( shadowMapping )
{
	// TODO: Dynamically resize queue if it gets too small instead of
	// using a big conservative value
	m_dJobQueue.resize( 10000000 );

	m_dDepthBuffer.resize( resolution() );
	m_dVoxelBuffer.resize( resolution() );
	m_dShadowMap.resize( resolution() );

	tDiffuse.normalized = true;
	tDiffuse.filterMode = hipFilterModeLinear;
	tDiffuse.addressMode[0] = tDiffuse.addressMode[1] = hipAddressModeWrap;

	tIllum.normalized = true;
	tIllum.filterMode = hipFilterModeLinear;
	tIllum.addressMode[0] = tIllum.addressMode[1] = hipAddressModeWrap;

	tSpec.normalized = true;
	tSpec.filterMode = hipFilterModeLinear;
	tSpec.addressMode[0] = tSpec.addressMode[1] = hipAddressModeWrap;

	tNormal.normalized = true;
	tNormal.filterMode = hipFilterModeLinear;
	tNormal.addressMode[0] = tNormal.addressMode[1] = hipAddressModeWrap;
}



void Renderer::render
(
	Object3d & obj,
	Camera const & cam,
	Light const & light,

	uchar4 * outColorBuffer
)
{
	int animationFrameIndex = BFSOctreeUpdate( & obj.data );

	if( m_shadowMapping )
	{
		fillJobQueue( thrust::raw_pointer_cast( obj.data.d_jobs->data() ), obj.data.jobCount );
		clearDepthBuffer();
		clearShadowMap();

		rasterize
		( 
			obj, 
			light.camera(),
			light,
			animationFrameIndex,
		
			true,

			outColorBuffer
		);
	}

	fillJobQueue( thrust::raw_pointer_cast( obj.data.d_jobs->data() ), obj.data.jobCount );
	clearColorBuffer( outColorBuffer );
	clearDepthBuffer();
	if( ! m_shadowMapping )
	{
		clearShadowMap();
	}

	rasterize
	( 
		obj, 
		cam,
		light,
		animationFrameIndex,
		
		false,

		outColorBuffer
	);
}



void Renderer::rasterize
(
	Object3d const & obj,
	Camera const & cam,
	Light const & light,
	int animationFrameIndex,

	bool shadowPass,
		
	uchar4 * outColorBuffer
)
{
	int hStartIndex = 0;
	int hEndIndex = obj.data.jobCount;

	// TODO: Extract into DeviceQueue class
	thrust::device_vector< int > dStartIndex( 1 );
	thrust::device_vector< int > dEndIndex( 1 );
	thrust::device_vector< int > dTravQueuePtr( 1 );

	dTravQueuePtr[ 0 ] = hEndIndex;

	int octreeLevel = obj.data.level;
	do
	{		
		dStartIndex[ 0 ] = hStartIndex;
		dEndIndex[ 0 ] = hEndIndex;

		traverse<<< nBlocks( hEndIndex - hStartIndex, nTHREADS_TRAV_KERNEL ), nTHREADS_TRAV_KERNEL >>>
		(
			obj.data.innerNodeCount,
			obj.data.d_innerNodes,
			obj.data.d_leaves,
			obj.data.dim,
			obj.transform, cam.position(), cam.viewMatrix(), cam.projectionMatrix(),
			obj.data.d_animation, obj.data.boneCount,
			thrust::raw_pointer_cast( m_dDepthBuffer.data() ), thrust::raw_pointer_cast( m_dVoxelBuffer.data() ),
			m_frameWidth, m_frameHeight,
			animationFrameIndex,
			octreeLevel,
			thrust::raw_pointer_cast( dStartIndex.data() ), thrust::raw_pointer_cast( dEndIndex.data() ),
			thrust::raw_pointer_cast( dTravQueuePtr.data() ),
			thrust::raw_pointer_cast( m_dJobQueue.data() )
		);
		
		hStartIndex = hEndIndex;		
		hEndIndex = dTravQueuePtr[ 0 ];
		octreeLevel++;
	}
	while( hEndIndex - hStartIndex > 0 );
	
	hipBindTexture
	(
		(size_t *) 0,
		tDepthBuffer,
		(void *) thrust::raw_pointer_cast( m_dDepthBuffer.data() ),
		hipCreateChannelDesc< unsigned >(),
		(size_t) ( resolution() * sizeof( unsigned int ) )
	);
	if( shadowPass )
	{
		drawShadowMap<<< nBlocks( resolution(), nTHREADS_DRAW_SHADOW_KERNEL ), nTHREADS_DRAW_SHADOW_KERNEL >>>
		(
			thrust::raw_pointer_cast( m_dDepthBuffer.data() ), 
			thrust::raw_pointer_cast( m_dShadowMap.data() ), 
			thrust::raw_pointer_cast( m_dVoxelBuffer.data() ),
			m_frameWidth, m_frameHeight
		);
	}
	else
	{
		hipBindTextureToArray( tDiffuse, obj.data.diffuse->data(), hipCreateChannelDesc< uchar4 >() );
		hipBindTextureToArray( tIllum, obj.data.illum->data(), hipCreateChannelDesc< uchar4 >() );
		hipBindTextureToArray( tSpec, obj.data.spec->data(), hipCreateChannelDesc< uchar4 >() );
		hipBindTextureToArray( tNormal, obj.data.normal->data(), hipCreateChannelDesc< uchar4 >() );

		draw<<< nBlocks( resolution(), nTHREADS_DRAW_KERNEL ), nTHREADS_DRAW_KERNEL >>>
		(
			thrust::raw_pointer_cast( m_dDepthBuffer.data() ),
			outColorBuffer,
			thrust::raw_pointer_cast( m_dVoxelBuffer.data() ),
			thrust::raw_pointer_cast( m_dShadowMap.data() ),
			m_frameWidth, m_frameHeight,
			light.direction(),
			light.camera().viewProjectionMatrix(),
			light.diffusePower()
		);

		hipUnbindTexture( tDiffuse );
		hipUnbindTexture( tIllum );
		hipUnbindTexture( tSpec );
		hipUnbindTexture( tNormal );
	}
	hipUnbindTexture( tDepthBuffer );
}



void Renderer::clearColorBuffer( uchar4 * dpOutColorBuffer )
{
	uchar4 const colorBufferClearValue = make_uchar4( 51, 51, 51, 255 );
	thrust::device_ptr< uchar4 > wrappedPtr( dpOutColorBuffer );

	thrust::fill
	( 
		wrappedPtr,
		wrappedPtr + resolution(),
		colorBufferClearValue
	);
}

void Renderer::clearDepthBuffer()
{
	unsigned int const depthBufferClearValue = std::numeric_limits< unsigned int >::max();
	m_dDepthBuffer.assign( m_dDepthBuffer.size(), depthBufferClearValue );
}

void Renderer::clearShadowMap()
{
	float const shadowMapClearValue = 1;
	m_dShadowMap.assign( m_dShadowMap.size(), shadowMapClearValue );
}

void Renderer::fillJobQueue( BFSJob const * dpJobs, int jobCount )
{
	hipMemcpy
	(
		thrust::raw_pointer_cast( m_dJobQueue.data() ),
		dpJobs,
		jobCount * sizeof( BFSJob ),
		hipMemcpyDeviceToDevice
	);
}



int Renderer::resolution() const
{
	return m_frameWidth * m_frameHeight;
}



// static
int Renderer::nBlocks( int nElements, int nThreadsPerBlock )
{
	int result = nElements / nThreadsPerBlock;
	return result + ( result * nThreadsPerBlock < nElements );
}